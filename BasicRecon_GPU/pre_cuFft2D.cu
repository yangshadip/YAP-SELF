#include "hip/hip_runtime.h"
#include <cusp/complex.h>
#include <hipfft/hipfft.h>
#include <thrust\device_vector.h>
#include <thrust\device_ptr.h>
#include <hip/hip_runtime.h>
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

using namespace cusp;
using namespace thrust;

const unsigned int TILE_DIM = 32;
const unsigned int BLOCK_ROWS = 32;

__global__ void fftshift(complex<float> * d_idata, complex<float> * d_odata)
{
	__shared__ complex<float> tile[TILE_DIM][TILE_DIM + 1];
	int x = TILE_DIM * blockIdx.x + threadIdx.x;
	int y = TILE_DIM * blockIdx.y + threadIdx.y;
	int width = gridDim.x * TILE_DIM;
	tile[threadIdx.y][threadIdx.x] = d_idata[y*width + x];
	__syncthreads();
	if (blockIdx.x < width / (TILE_DIM * 2) && blockIdx.y < width / (TILE_DIM * 2))
	{
		d_odata[(y + width / 2)*width + x + width / 2] = tile[threadIdx.y][threadIdx.x];
	}
	if (blockIdx.x < width / (TILE_DIM * 2) && blockIdx.y >= width / (TILE_DIM * 2))
	{
		d_odata[(y - width / 2)*width + x + width / 2] = tile[threadIdx.y][threadIdx.x];
	}
	if (blockIdx.x >= width / (TILE_DIM * 2) && blockIdx.y < width / (TILE_DIM * 2))
	{
		d_odata[(y + width / 2)*width + x - width / 2] = tile[threadIdx.y][threadIdx.x];
	}
	if (blockIdx.x >= width / (TILE_DIM * 2) && blockIdx.y >= width / (TILE_DIM * 2))
	{
		d_odata[(y - width / 2)*width + x - width / 2] = tile[threadIdx.y][threadIdx.x];
	}
}

__global__  void ComplexScale(complex<float>* d_idata, 
	complex<float>* d_odata,
	float scale, 
	unsigned int width,
	unsigned int height)
{
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockIdx.y * blockDim.y;
	if (col< width && row < height)
	{
		d_odata[row*width + col] = d_idata[row*width + col] * scale;
	}
}

extern "C" void pre_cuFft2D(std::complex<float> * h_kspace, 
	std::complex<float> * h_image, 
	bool fft_forward, 
	unsigned int width, 
	unsigned int height)
{
	complex<float> *d_kspace, *d_image;
	device_vector<complex<float>> d_buffer(width * height);
	size_t lpitch;
	hipfftHandle plan;
	checkCudaErrors(hipfftPlan2d(&plan, height, width, HIPFFT_C2C));
	checkCudaErrors(hipMallocPitch((void **)&d_kspace, &lpitch, width * sizeof(complex<float>), height));
	checkCudaErrors(hipMallocPitch((void **)&d_image, &lpitch, width * sizeof(complex<float>), height));
	checkCudaErrors(hipMemcpy2D((void *)d_kspace, lpitch, (void *)h_kspace, sizeof(float)*2*width, sizeof(float)*2*width, height, hipMemcpyHostToDevice));
	dim3 dimBlock(width, 1, 1);
	dim3 dimGrid(1, height, 1);
	dim3 grid(width / TILE_DIM, height / TILE_DIM, 1);
	dim3 block(TILE_DIM, BLOCK_ROWS, 1);

	fftshift <<< grid, block >>>(d_kspace, raw_pointer_cast(d_buffer.data()));

	if (fft_forward)
	{
		checkCudaErrors(hipfftExecC2C(plan, raw_pointer_cast(d_buffer.data()), d_image, HIPFFT_FORWARD));
	}
	else
	{
		checkCudaErrors(hipfftExecC2C(plan, raw_pointer_cast(d_buffer.data()), d_image, HIPFFT_BACKWARD));
	}
	fftshift << <grid, block >> >(d_image, raw_pointer_cast(d_buffer.data()));
	float scale = 1 / (float)std::sqrt(width * height);
	ComplexScale << <dimGrid, dimBlock >> >(raw_pointer_cast(d_buffer.data()), d_image, scale, width, height);
	checkCudaErrors(hipMemcpy2D((void *)h_image, lpitch, (void *)d_image, sizeof(float)*2*width, sizeof(float)*2*width, height, hipMemcpyDeviceToHost));
	checkCudaErrors(hipfftDestroy(plan));
	checkCudaErrors(hipFree(d_kspace));
	checkCudaErrors(hipFree(d_image));
}